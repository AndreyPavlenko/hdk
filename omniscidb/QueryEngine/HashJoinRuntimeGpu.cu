#include "hip/hip_runtime.h"
/*
 * Copyright 2017 MapD Technologies, Inc.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "HashJoinRuntime.cpp"

__global__ void fill_hash_join_buff_wrapper(int32_t* buff,
                                            const int32_t invalid_slot_val,
                                            const JoinColumn join_column,
                                            const JoinColumnTypeInfo type_info,
                                            int* err) {
  int partial_err = SUFFIX(fill_hash_join_buff)(buff, invalid_slot_val, join_column, type_info, NULL, NULL, -1, -1);
  atomicCAS(err, 0, partial_err);
}

void fill_hash_join_buff_on_device(int32_t* buff,
                                   const int32_t invalid_slot_val,
                                   int* dev_err_buff,
                                   const JoinColumn join_column,
                                   const JoinColumnTypeInfo type_info,
                                   const size_t block_size_x,
                                   const size_t grid_size_x) {
  fill_hash_join_buff_wrapper<<<grid_size_x, block_size_x>>>(
      buff, invalid_slot_val, join_column, type_info, dev_err_buff);
}

__global__ void fill_hash_join_buff_wrapper_sharded(int32_t* buff,
                                                    const int32_t invalid_slot_val,
                                                    const JoinColumn join_column,
                                                    const JoinColumnTypeInfo type_info,
                                                    const ShardInfo shard_info,
                                                    int* err) {
  int partial_err = SUFFIX(fill_hash_join_buff_sharded)(
      buff, invalid_slot_val, join_column, type_info, shard_info, NULL, NULL, -1, -1);
  atomicCAS(err, 0, partial_err);
}

void fill_hash_join_buff_on_device_sharded(int32_t* buff,
                                           const int32_t invalid_slot_val,
                                           int* dev_err_buff,
                                           const JoinColumn join_column,
                                           const JoinColumnTypeInfo type_info,
                                           const ShardInfo shard_info,
                                           const size_t block_size_x,
                                           const size_t grid_size_x) {
  fill_hash_join_buff_wrapper_sharded<<<grid_size_x, block_size_x>>>(
      buff, invalid_slot_val, join_column, type_info, shard_info, dev_err_buff);
}

__global__ void init_hash_join_buff_wrapper(int32_t* buff,
                                            const int32_t hash_entry_count,
                                            const int32_t invalid_slot_val) {
  SUFFIX(init_hash_join_buff)(buff, hash_entry_count, invalid_slot_val, -1, -1);
}

void init_hash_join_buff_on_device(int32_t* buff,
                                   const int32_t hash_entry_count,
                                   const int32_t invalid_slot_val,
                                   const size_t block_size_x,
                                   const size_t grid_size_x) {
  init_hash_join_buff_wrapper<<<grid_size_x, block_size_x>>>(buff, hash_entry_count, invalid_slot_val);
}
